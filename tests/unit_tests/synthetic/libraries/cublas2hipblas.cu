// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"

int main() {
  printf("14. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: hipblasOperation_t blasOperation;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: hipblasStatus_t blasStatus;
  // CHECK-NEXT: hipblasStatus_t blasStatus_t;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: hipblasFillMode_t blasFillMode;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: hipblasDiagType_t blasDiagType;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: hipblasSideMode_t blasSideMode;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: hipblasPointerMode_t blasPointerMode;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: hipblasAtomicsMode_t blasAtomicsMode;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: hipblasHandle_t blasHandle;
  hipblasHandle_t blasHandle;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t* mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t* atomics_mode);
  // CHECK: blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);
  blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t atomics_mode);
  // CHECK: blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);
  blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCreate_v2(cublasHandle_t* handle);
  // CUDA: #define cublasCreate cublasCreate_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCreate(hipblasHandle_t* handle);
  // CHECK: blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDestroy_v2(cublasHandle_t handle);
  // CUDA: #define cublasDestroy cublasDestroy_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDestroy(hipblasHandle_t handle);
  // CHECK: blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetStream_v2(cublasHandle_t handle, cudaStream_t streamId);
  // CUDA: #define cublasSetStream cublasSetStream_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetStream(hipblasHandle_t handle, hipStream_t streamId);
  // CHECK: blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetStream_v2(cublasHandle_t handle, cudaStream_t* streamId);
  // CUDA: #define cublasGetStream cublasGetStream_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetStream(hipblasHandle_t handle, hipStream_t* streamId);
  // CHECK: blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t mode);
  // CUDA: #define cublasSetPointerMode cublasSetPointerMode_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t mode);
  // CHECK: blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t* mode);
  // CUDA: #define cublasGetPointerMode cublasGetPointerMode_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t* mode);
  // CHECK: blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);

  int n = 0;
  int nrhs = 0;
  int m = 0;
  int num = 0;
  int lda = 0;
  int ldb = 0;
  int ldc = 0;
  int res = 0;
  int incx = 0;
  int incy = 0;
  int k = 0;
  int kl = 0;
  int ku = 0;
  int batchCount = 0;
  int P = 0;
  int info = 0;
  void* image = nullptr;
  void* image_2 = nullptr;
  void* deviceptr = nullptr;

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVector(int n, int elemSize, const void* x, int incx, void* devicePtr, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // CHECK: blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // CHECK: blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVectorAsync(int n, int elemSize, const void* hostPtr, int incx, void* devicePtr, int incy, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetVectorAsync(int n, int elemSize, const void* x, int incx, void* y, int incy, hipStream_t stream);
  // CHECK: blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVectorAsync(int n, int elemSize, const void* devicePtr, int incx, void* hostPtr, int incy, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetVectorAsync(int n, int elemSize, const void* x, int incx, void* y, int incy, hipStream_t stream);
  // CHECK: blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);

  int rows = 0;
  int cols = 0;

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetMatrix(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb);
  // CHECK: blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetMatrix(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb);
  // CHECK: blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetMatrixAsync(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb, hipStream_t stream);
  // CHECK: blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetMatrixAsync(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb, hipStream_t stream);
  // CHECK: blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  float fa = 0;
  float fA = 0;
  float fb = 0;
  float fB = 0;
  float fx = 0;
  float fx1 = 0;
  float fy = 0;
  float fy1 = 0;
  float fc = 0;
  float fC = 0;
  float fs = 0;
  float fd1 = 0;
  float fd2 = 0;
  float fresult = 0;

  float** fAarray = 0;
  const float** const fAarray_const = const_cast<const float**>(fAarray);
  float** fBarray = 0;
  const float** const fBarray_const = const_cast<const float**>(fBarray);
  float** fCarray = 0;
  float** fTauarray = 0;

  double da = 0;
  double dA = 0;
  double db = 0;
  double dB = 0;
  double dx = 0;
  double dx1 = 0;
  double dy = 0;
  double dy1 = 0;
  double dc = 0;
  double dC = 0;
  double ds = 0;
  double dd1 = 0;
  double dd2 = 0;
  double dresult = 0;

  double** dAarray = 0;
  const double** const dAarray_const = const_cast<const double**>(dAarray);
  double** dBarray = 0;
  const double** const dBarray_const = const_cast<const double**>(dBarray);
  double** dCarray = 0;
  double** dTauarray = 0;

  void** voidAarray = nullptr;
  const void** const voidAarray_const = const_cast<const void**>(voidAarray);
  void** voidBarray = nullptr;
  const void** const voidBarray_const = const_cast<const void**>(voidBarray);
  void** voidCarray = nullptr;

  // NOTE: float CUBLASWINAPI cublasSnrm2(int n, const float* x, int incx) is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSnrm2_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSnrm2(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // CHECK: blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);

  // NOTE: double CUBLASWINAPI cublasDnrm2(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDnrm2_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDnrm2(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);

  // CHECK: hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  // CHECK: hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;
  hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;

  // CHECK: hipComplex** complexAarray = 0;
  // CHECK: const hipComplex** const complexAarray_const = const_cast<const hipComplex**>(complexAarray);
  // CHECK-NEXT: hipComplex** complexBarray = 0;
  // CHECK: const hipComplex** const complexBarray_const = const_cast<const hipComplex**>(complexBarray);
  // CHECK-NEXT: hipComplex** complexCarray = 0;
  // CHECK-NEXT: hipComplex** complexTauarray = 0;
  // CHECK-NEXT: hipComplex** complexXarray = 0;
  // CHECK: const hipComplex** const complexXarray_const = const_cast<const hipComplex**>(complexXarray);
  // CHECK-NEXT: hipComplex** complexYarray = 0;
  // CHECK: const hipComplex** const complexYarray_const = const_cast<const hipComplex**>(complexYarray);
  hipComplex** complexAarray = 0;
  const hipComplex** const complexAarray_const = const_cast<const hipComplex**>(complexAarray);
  hipComplex** complexBarray = 0;
  const hipComplex** const complexBarray_const = const_cast<const hipComplex**>(complexBarray);
  hipComplex** complexCarray = 0;
  hipComplex** complexTauarray = 0;
  hipComplex** complexXarray = 0;
  const hipComplex** const complexXarray_const = const_cast<const hipComplex**>(complexXarray);
  hipComplex** complexYarray = 0;
  const hipComplex** const complexYarray_const = const_cast<const hipComplex**>(complexYarray);

  // CHECK: hipDoubleComplex** dcomplexAarray = 0;
  // CHECK: const hipDoubleComplex** const dcomplexAarray_const = const_cast<const hipDoubleComplex**>(dcomplexAarray);
  // CHECK-NEXT: hipDoubleComplex** dcomplexBarray = 0;
  // CHECK: const hipDoubleComplex** const dcomplexBarray_const = const_cast<const hipDoubleComplex**>(dcomplexBarray);
  // CHECK-NEXT: hipDoubleComplex** dcomplexCarray = 0;
  // CHECK-NEXT: hipDoubleComplex** dcomplexTauarray = 0;
  // CHECK-NEXT: hipDoubleComplex** dcomplexXarray = 0;
  // CHECK: const hipDoubleComplex** const dcomplexXarray_const = const_cast<const hipDoubleComplex**>(dcomplexXarray);
  // CHECK-NEXT: hipDoubleComplex** dcomplexYarray = 0;
  // CHECK: const hipDoubleComplex** const dcomplexYarray_const = const_cast<const hipDoubleComplex**>(dcomplexYarray);
  hipDoubleComplex** dcomplexAarray = 0;
  const hipDoubleComplex** const dcomplexAarray_const = const_cast<const hipDoubleComplex**>(dcomplexAarray);
  hipDoubleComplex** dcomplexBarray = 0;
  const hipDoubleComplex** const dcomplexBarray_const = const_cast<const hipDoubleComplex**>(dcomplexBarray);
  hipDoubleComplex** dcomplexCarray = 0;
  hipDoubleComplex** dcomplexTauarray = 0;
  hipDoubleComplex** dcomplexXarray = 0;
  const hipDoubleComplex** const dcomplexXarray_const = const_cast<const hipDoubleComplex**>(dcomplexXarray);
  hipDoubleComplex** dcomplexYarray = 0;
  const hipDoubleComplex** const dcomplexYarray_const = const_cast<const hipDoubleComplex**>(dcomplexYarray);

  // NOTE: float CUBLASWINAPI cublasScnrm2(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScnrm2_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScnrm2_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, float* result);
  // CHECK: blasStatus = hipblasScnrm2_v2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);

  // NOTE: double CUBLASWINAPI cublasDznrm2(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDznrm2_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDznrm2_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDznrm2_v2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);

  // NOTE: float CUBLASWINAPI cublasSdot(int n, const float* x, int incx, const float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdot_v2(cublasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSdot(hipblasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // CHECK: blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // NOTE: double CUBLASWINAPI cublasDdot(int n, const double* x, int incx, const double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdot_v2(cublasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDdot(hipblasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // CHECK: blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // NOTE: cuComplex CUBLASWINAPI cublasCdotu(int n, const cuComplex* x, int incx, const cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotu_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotu_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* result);
  // CHECK: blasStatus = hipblasCdotu_v2(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // NOTE: cuComplex CUBLASWINAPI cublasCdotc(int n, const cuComplex* x, int incx, const cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotc_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotc_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* result);
  // CHECK: blasStatus = hipblasCdotc_v2(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // NOTE: cuDoubleComplex CUBLASWINAPI cublasZdotu(int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotu_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotu_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotu_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // NOTE: cuDoubleComplex CUBLASWINAPI cublasZdotc(int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotc_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotc_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotc_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // NOTE: void CUBLASWINAPI cublasSscal(int n, float alpha, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSscal_v2(cublasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSscal(hipblasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // CHECK: blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDscal(int n, double alpha, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDscal_v2(cublasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDscal(hipblasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // CHECK: blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);

  // NOTE: void CUBLASWINAPI cublasCscal(int n, cuComplex alpha, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCscal_v2(cublasHandle_t handle, int n, const cuComplex* alpha, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCscal_v2(hipblasHandle_t handle, int n, const hipComplex* alpha, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCscal_v2(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasCsscal(int n, float alpha, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsscal_v2(cublasHandle_t handle, int n, const float* alpha, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsscal_v2(hipblasHandle_t handle, int n, const float* alpha, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCsscal_v2(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZscal(int n, cuDoubleComplex alpha, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZscal_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZscal_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZscal_v2(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasZdscal(int n, double alpha, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdscal_v2(cublasHandle_t handle, int n, const double* alpha, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdscal_v2(hipblasHandle_t handle, int n, const double* alpha, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZdscal_v2(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasSaxpy(int n, float alpha, const float* x, int incx, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSaxpy_v2(cublasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSaxpy(hipblasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDaxpy(int n, double alpha, const double* x, int incx, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDaxpy_v2(cublasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDaxpy(hipblasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCaxpy(int n, cuComplex alpha, const cuComplex* x, int incx, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCaxpy_v2(cublasHandle_t handle, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCaxpy_v2(hipblasHandle_t handle, int n, const hipComplex* alpha, const hipComplex* x, int incx, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCaxpy_v2(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZaxpy(int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZaxpy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZaxpy_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZaxpy_v2(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasScopy(int n, const float* x, int incx, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScopy_v2(cublasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScopy(hipblasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDcopy(int n, const double* x, int incx, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDcopy_v2(cublasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDcopy(hipblasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCcopy(int n, const cuComplex* x, int incx, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCcopy_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCcopy_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCcopy_v2(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZcopy(int n, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZcopy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZcopy_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZcopy_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSswap(int n, float* x, int incx, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSswap_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSswap(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDswap(int n, double* x, int incx, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDswap_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDswap(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCswap(int n, cuComplex* x, int incx, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCswap_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCswap_v2(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCswap_v2(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZswap(int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZswap_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZswap_v2(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZswap_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // NOTE: int CUBLASWINAPI cublasIsamax(int n, const float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamax_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamax(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIdamax(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamax_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamax(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIcamax(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamax_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamax_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIcamax_v2(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIzamax(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamax_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamax_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIzamax_v2(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIsamin(int n, const float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamin_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamin(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIdamin(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamin_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamin(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIcamin(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamin_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamin_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIcamin_v2(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIzamin(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamin_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamin_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIzamin_v2(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);

  // NOTE: float CUBLASWINAPI cublasSasum(int n, const float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSasum_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSasum(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // CHECK: blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);

  // NOTE: double CUBLASWINAPI cublasDasum(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDasum_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDasum(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);

  // NOTE: float CUBLASWINAPI cublasScasum(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScasum_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScasum_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, float* result);
  // CHECK: blasStatus = hipblasScasum_v2(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);

  // NOTE: double CUBLASWINAPI cublasDzasum(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDzasum_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDzasum_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDzasum_v2(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);

  // NOTE: void CUBLASWINAPI cublasSrot(int n, float* x, int incx, float* y, int incy, float sc, float ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrot_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrot(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);

  // NOTE: void CUBLASWINAPI cublasDrot(int n, double* x, int incx, double* y, int incy, double sc, double ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrot_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrot(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);

  // NOTE: void CUBLASWINAPI cublasCrot(int n, cuComplex* x, int incx, cuComplex* y, int incy, float c, cuComplex s); is not supported by HIP
  // CUDA: CUBLASAPI CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const cuComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCrot_v2(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy, const float* c, const hipComplex* s);
  // CHECK: blasStatus = hipblasCrot_v2(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);

  // NOTE: void CUBLASWINAPI cublasCsrot(int n, cuComplex* x, int incx, cuComplex* y, int incy, float c, float s); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsrot_v2(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasCsrot_v2(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);

  // NOTE: void CUBLASWINAPI cublasZrot(int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, double sc, cuDoubleComplex cs); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const cuDoubleComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZrot_v2(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy, const double* c, const hipDoubleComplex* s);
  // CHECK: blasStatus = hipblasZrot_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);

  // NOTE: void CUBLASWINAPI cublasZdrot(int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, double c, double s); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdrot_v2(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasZdrot_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);

  // NOTE: void CUBLASWINAPI cublasSrotg(float* sa, float* sb, float* sc, float* ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotg_v2(cublasHandle_t handle, float* a, float* b, float* c, float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotg(hipblasHandle_t handle, float* a, float* b, float* c, float* s);
  // CHECK: blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);
  blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);

  // NOTE: void CUBLASWINAPI cublasDrotg(double* sa, double* sb, double* sc, double* ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotg_v2(cublasHandle_t handle, double* a, double* b, double* c, double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotg(hipblasHandle_t handle, double* a, double* b, double* c, double* s);
  // CHECK: blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);
  blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);

  // NOTE: void CUBLASWINAPI cublasCrotg(cuComplex* ca, cuComplex cb, float* sc, cuComplex* cs); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrotg_v2(cublasHandle_t handle, cuComplex* a, cuComplex* b, float* c, cuComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCrotg_v2(hipblasHandle_t handle, hipComplex* a, hipComplex* b, float* c, hipComplex* s);
  // CHECK: blasStatus = hipblasCrotg_v2(blasHandle, &complexa, &complexb, &fc, &complexs);
  blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);

  // NOTE: void CUBLASWINAPI cublasZrotg(cuDoubleComplex* ca, cuDoubleComplex cb, double* sc, cuDoubleComplex* cs); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrotg_v2(cublasHandle_t handle, cuDoubleComplex* a, cuDoubleComplex* b, double* c, cuDoubleComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZrotg_v2(hipblasHandle_t handle, hipDoubleComplex* a, hipDoubleComplex* b, double* c, hipDoubleComplex* s);
  // CHECK: blasStatus = hipblasZrotg_v2(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);

  // NOTE: void CUBLASWINAPI cublasSrotm(int n, float* x, int incx, float* y, int incy, const float* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotm_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotm(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // CHECK: blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // NOTE: void CUBLASWINAPI cublasDrotm(int n, double* x, int incx, double* y, int incy, const double* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotm_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotm(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // CHECK: blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // NOTE: void CUBLASWINAPI cublasSrotmg(float* sd1, float* sd2, float* sx1, const float* sy1, float* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotmg_v2(cublasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotmg(hipblasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // CHECK: blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);

  // NOTE: void CUBLASWINAPI cublasDrotmg(double* sd1, double* sd2, double* sx1, const double* sy1, double* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotmg_v2(cublasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotmg(hipblasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // CHECK: blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);

  // NOTE: void CUBLASWINAPI cublasSgemv(char trans, int m, int n, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDgemv(char trans, int m, int n, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCgemv(char trans, int m, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemv_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCgemv_v2(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZgemv(char trans, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemv_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZgemv_v2(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSgbmv(char trans, int m, int n, int kl, int ku, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDgbmv(char trans, int m, int n, int kl, int ku, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCgbmv(char trans, int m, int n, int kl, int ku, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgbmv_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCgbmv_v2(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZgbmv(char trans, int m, int n, int kl, int ku, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgbmv_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZgbmv_v2(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasStrmv(char uplo, char trans, char diag, int n, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtrmv(char uplo, char trans, char diag, int n, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtrmv(char uplo, char trans, char diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipComplex* AP, int lda, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtrmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtrmv(char uplo, char trans, char diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipDoubleComplex* AP, int lda, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtrmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStbmv(char uplo, char trans, char diag, int n, int k, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtbmv(char uplo, char trans, char diag, int n, int k, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtbmv(char uplo, char trans, char diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtbmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const hipComplex* AP, int lda, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtbmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtbmv(char uplo, char trans, char diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtbmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const hipDoubleComplex* AP, int lda, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtbmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStpmv(char uplo, char trans, char diag, int n, const float* AP, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, float* x, int incx);
  // CHECK: blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtpmv(char uplo, char trans, char diag, int n, const double* AP, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, double* x, int incx);
  // CHECK: blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtpmv(char uplo, char trans, char diag, int n, const cuComplex* AP, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtpmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipComplex* AP, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtpmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtpmv(char uplo, char trans, char diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtpmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipDoubleComplex* AP, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtpmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStrsv(char uplo, char trans, char diag, int n, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtrsv(char uplo, char trans, char diag, int n, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtrsv(char uplo, char trans, char diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipComplex* AP, int lda, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtrsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtrsv(char uplo, char trans, char diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipDoubleComplex* AP, int lda, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtrsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStpsv(char uplo, char trans, char diag, int n, const float* AP, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, float* x, int incx);
  // CHECK: blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtpsv(char uplo, char trans, char diag, int n, const double* AP, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, double* x, int incx);
  // CHECK: blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtpsv(char uplo, char trans, char diag, int n, const cuComplex* AP, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtpsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipComplex* AP, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtpsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtpsv(char uplo, char trans, char diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtpsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipDoubleComplex* AP, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtpsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStbsv(char uplo, char trans, char diag, int n, int k, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtbsv(char uplo, char trans, char diag, int n, int k, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtbsv(char uplo, char trans, char diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtbsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const hipComplex* AP, int lda, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtbsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtbsv(char uplo, char trans, char diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtbsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const hipDoubleComplex* AP, int lda, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtbsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasSsymv(char uplo, int n, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDsymv(char uplo, int n, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsymv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCsymv_v2(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsymv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZsymv_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasChemv(char uplo, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChemv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasChemv_v2(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZhemv(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhemv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhemv_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSsbmv(char uplo, int n, int k, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDsbmv(char uplo, int n, int k, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasChbmv(char uplo, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChbmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasChbmv_v2(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZhbmv(char uplo, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhbmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhbmv_v2(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSspmv(char uplo, int n, float alpha, const float* AP, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDspmv(char uplo, int n, double alpha, const double* AP, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasChpmv(char uplo, int n, cuComplex alpha, const cuComplex* AP, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* AP, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* AP, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasChpmv_v2(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZhpmv(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* AP, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* AP, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhpmv_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSger(int m, int n, float alpha, const float* x, int incx, const float* y, int incy, float* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSger_v2(cublasHandle_t handle, int m, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSger(hipblasHandle_t handle, int m, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP, int lda);
  // CHECK: blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // NOTE: void CUBLASWINAPI cublasDger(int m, int n, double alpha, const double* x, int incx, const double* y, int incy, double* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDger_v2(cublasHandle_t handle, int m, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDger(hipblasHandle_t handle, int m, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP, int lda);
  // CHECK: blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // NOTE: void CUBLASWINAPI cublasCgeru(int m, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeru_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgeru_v2(hipblasHandle_t handle, int m, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCgeru_v2(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // NOTE: void CUBLASWINAPI cublasCgerc(int m, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgerc_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgerc_v2(hipblasHandle_t handle, int m, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCgerc_v2(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // NOTE: void CUBLASWINAPI cublasZgeru(int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeru_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgeru_v2(hipblasHandle_t handle, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZgeru_v2(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasZgerc(int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgerc_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgerc_v2(hipblasHandle_t handle, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZgerc_v2(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasSsyr(char uplo, int n, float alpha, const float* x, int incx, float* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP, int lda);
  // CHECK: blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);
  blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);

  // NOTE: void CUBLASWINAPI cublasDsyr(char uplo, int n, double alpha, const double* x, int incx, double* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP, int lda);
  // CHECK: blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);
  blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyr_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* x, int incx, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCsyr_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCsyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyr_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZsyr_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasCher(char uplo, int n, float alpha, const cuComplex* x, int incx, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCher_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const hipComplex* x, int incx, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCher_v2(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);

  // NOTE: void CUBLASWINAPI cublasZher(char uplo, int n, double alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZher_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZher_v2(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasSspr(char uplo, int n, float alpha, const float* x, int incx, float* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP);
  // CHECK: blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);
  blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);

  // NOTE: void CUBLASWINAPI cublasDspr(char uplo, int n, double alpha, const double* x, int incx, double* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, double* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, double* AP);
  // CHECK: blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);
  blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);

  // NOTE: void CUBLASWINAPI cublasChpr(char uplo, int n, float alpha, const cuComplex* x, int incx, cuComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpr_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const hipComplex* x, int incx, hipComplex* AP);
  // CHECK: blasStatus = hipblasChpr_v2(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);
  blasStatus = hipblasChpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);

  // NOTE: void CUBLASWINAPI cublasZhpr(char uplo, int n, double alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpr_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* AP);
  // CHECK: blasStatus = hipblasZhpr_v2(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);
  blasStatus = hipblasZhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);

  // NOTE: void CUBLASWINAPI cublasSsyr2(char uplo, int n, float alpha, const float* x, int incx, const float* y, int incy, float* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP, int lda);
  // CHECK: blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // NOTE: void CUBLASWINAPI cublasDsyr2(char uplo, int n, double alpha, const double* x, int incx, const double* y, int incy, double* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP, int lda);
  // CHECK: blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyr2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCsyr2_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCsyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyr2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZsyr2_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasCher2(char uplo, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCher2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCher2_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // NOTE: void CUBLASWINAPI cublasZher2(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZher2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZher2_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasSspr2(char uplo, int n, float alpha, const float* x, int incx, const float* y, int incy, float* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP);
  // CHECK: blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);
  blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);

  // NOTE: void CUBLASWINAPI cublasDspr2(char uplo, int n, double alpha, const double* x, int incx, const double* y, int incy, double* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP);
  // CHECK: blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);
  blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);

  // NOTE: void CUBLASWINAPI cublasChpr2(char uplo, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpr2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP);
  // CHECK: blasStatus = hipblasChpr2_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);
  blasStatus = hipblasChpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);

  // NOTE: void CUBLASWINAPI cublasZhpr2(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpr2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP);
  // CHECK: blasStatus = hipblasZhpr2_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);
  blasStatus = hipblasZhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);

  hipblasOperation_t transa, transb;

  // NOTE: void CUBLASWINAPI cublasSgemm(char transa, char transb, int m, int n, int k, float alpha, const float* A, int lda, const float* B, int ldb, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // NOTE: void CUBLASWINAPI cublasDgemm(char transa, char transb, int m, int n, int k, double alpha, const double* A, int lda, const double* B, int ldb, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // NOTE: void CUBLASWINAPI cublasCgemm(char transa, char transb, int m, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemm_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCgemm_v2(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZgemm(char transa, char transb, int m, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemm_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZgemm_v2(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* const Aarray[], int lda, const float* const Barray[], int ldb, const float* beta, float* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const float* alpha, const float* const AP[], int lda, const float* const BP[], int ldb, const float* beta, float* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasSgemmBatched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);
  blasStatus = hipblasSgemmBatched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* const Aarray[], int lda, const double* const Barray[], int ldb, const double* beta, double* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const double* alpha, const double* const AP[], int lda, const double* const BP[], int ldb, const double* beta, double* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasDgemmBatched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);
  blasStatus = hipblasDgemmBatched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* const Aarray[], int lda, const cuComplex* const Barray[], int ldb, const cuComplex* beta, cuComplex* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemmBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipComplex* alpha, const hipComplex* const AP[], int lda, const hipComplex* const BP[], int ldb, const hipComplex* beta, hipComplex* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasCgemmBatched_v2(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);
  blasStatus = hipblasCgemmBatched(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* const Aarray[], int lda, const cuDoubleComplex* const Barray[], int ldb, const cuDoubleComplex* beta, cuDoubleComplex* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemmBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* const AP[], int lda, const hipDoubleComplex* const BP[], int ldb, const hipDoubleComplex* beta, hipDoubleComplex* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasZgemmBatched_v2(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);
  blasStatus = hipblasZgemmBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);

  // NOTE: void CUBLASWINAPI cublasSsyrk(char uplo, char trans, int n, int k, float alpha, const float* A, int lda, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyrk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const float* AP, int lda, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);
  blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);

  // NOTE: void CUBLASWINAPI cublasDsyrk(char uplo, char trans, int n, int k, double alpha, const double* A, int lda, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyrk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const double* AP, int lda, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);
  blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);

  // NOTE: void CUBLASWINAPI cublasCsyrk(char uplo, char trans, int n, int k, cuComplex alpha, const cuComplex* A, int lda, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyrk_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsyrk_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZsyrk(char uplo, char trans, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyrk_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsyrk_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasCherk(char uplo, char trans, int n, int k, float alpha, const cuComplex* A, int lda, float beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const cuComplex* A, int lda, const float* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCherk_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const hipComplex* AP, int lda, const float* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCherk_v2(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);
  blasStatus = hipblasCherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZherk(char uplo, char trans, int n, int k, double alpha, const cuDoubleComplex* A, int lda, double beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const cuDoubleComplex* A, int lda, const double* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZherk_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const hipDoubleComplex* AP, int lda, const double* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZherk_v2(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasSsyr2k(char uplo, char trans, int n, int k, float alpha, const float* A, int lda, const float* B, int ldb, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyr2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);

  // NOTE: void CUBLASWINAPI cublasDsyr2k(char uplo, char trans, int n, int k, double alpha, const double* A, int lda, const double* B, int ldb, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyr2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);

  // NOTE: void CUBLASWINAPI cublasCsyr2k(char uplo, char trans, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyr2k_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsyr2k_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZsyr2k(char uplo, char trans, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyr2k_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsyr2k_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyrkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyrkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyrkx_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsyrkx_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyrkx_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsyrkx_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasCher2k(char uplo, char trans, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, float beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCher2k_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const float* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCher2k_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZher2k(char uplo, char trans, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, double beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZher2k_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const double* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZher2k_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCherkx_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const float* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCherkx_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCherkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZherkx_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const double* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZherkx_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasSsymm(char side, char uplo, int m, int n, float alpha, const float* A, int lda, const float* B, int ldb, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsymm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // NOTE: void CUBLASWINAPI cublasDsymm(char side, char uplo, int m, int n, double alpha, const double* A, int lda, const double* B, int ldb, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsymm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // NOTE: void CUBLASWINAPI cublasCsymm(char side, char uplo, int m, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsymm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsymm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZsymm(char side, char uplo, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsymm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsymm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasChemm(char side, char uplo, int m, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChemm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasChemm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasChemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZhemm(char side, char uplo, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhemm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZhemm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasStrsm(char side, char uplo, char transa, char diag, int m, int n, float alpha, const float* A, int lda, float* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, float* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const float* alpha, float* AP, int lda, float* BP, int ldb);
  // CHECK: blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);
  blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);

  // NOTE: void CUBLASWINAPI cublasDtrsm(char side, char uplo, char transa, char diag, int m, int n, double alpha, const double* A, int lda, double* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, double* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const double* alpha, double* AP, int lda, double* BP, int ldb);
  // CHECK: blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);
  blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);

  // NOTE: void CUBLASWINAPI cublasCtrsm(char side, char uplo, char transa, char diag, int m, int n, cuComplex alpha, const cuComplex* A, int lda, cuComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, cuComplex* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipComplex* alpha, const hipComplex* AP, int lda, hipComplex* BP, int ldb);
  // CHECK: blasStatus = hipblasCtrsm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);
  blasStatus = hipblasCtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);

  // NOTE: void CUBLASWINAPI cublasZtrsm(char side, char uplo, char transa, char diag, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, hipDoubleComplex* BP, int ldb);
  // CHECK: blasStatus = hipblasZtrsm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);
  blasStatus = hipblasZtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const float* alpha, const float* A, int lda, const float* beta, const float* B, int ldb, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgeam(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const float* alpha, const float* AP, int lda, const float* beta, const float* BP, int ldb, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);
  blasStatus = hipblasSgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const double* alpha, const double* A, int lda, const double* beta, const double* B, int ldb, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgeam(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const double* alpha, const double* AP, int lda, const double* beta, const double* BP, int ldb, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, const cuComplex* B, int ldb, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgeam_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* beta, const hipComplex* BP, int ldb, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCgeam_v2(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCgeam(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, const cuDoubleComplex* B, int ldb, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgeam_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* beta, const hipDoubleComplex* BP, int ldb, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZgeam_v2(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZgeam(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgetrfBatched(cublasHandle_t handle, int n, float* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgetrfBatched(hipblasHandle_t handle, const int n, float* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgetrfBatched(blasHandle, n, fAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasSgetrfBatched(blasHandle, n, fAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgetrfBatched(cublasHandle_t handle, int n, double* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgetrfBatched(hipblasHandle_t handle, const int n, double* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgetrfBatched(blasHandle, n, dAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasDgetrfBatched(blasHandle, n, dAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgetrfBatched(cublasHandle_t handle, int n, cuComplex* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgetrfBatched_v2(hipblasHandle_t handle, const int n, hipComplex* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgetrfBatched_v2(blasHandle, n, complexAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasCgetrfBatched(blasHandle, n, complexAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgetrfBatched(cublasHandle_t handle, int n, cuDoubleComplex* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgetrfBatched_v2(hipblasHandle_t handle, const int n, hipDoubleComplex* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgetrfBatched_v2(blasHandle, n, dcomplexAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasZgetrfBatched(blasHandle, n, dcomplexAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgetriBatched(cublasHandle_t handle, int n, const float* const A[], int lda, const int* P, float* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgetriBatched(hipblasHandle_t handle, const int n, float* const A[], const int lda, int* ipiv, float* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgetriBatched(blasHandle, n, fAarray_const, lda, &P, fCarray, ldc, &info, batchCount);
  blasStatus = hipblasSgetriBatched(blasHandle, n, fAarray_const, lda, &P, fCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgetriBatched(cublasHandle_t handle, int n, const double* const A[], int lda, const int* P, double* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgetriBatched(hipblasHandle_t handle, const int n, double* const A[], const int lda, int* ipiv, double* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgetriBatched(blasHandle, n, dAarray_const, lda, &P, dCarray, ldc, &info, batchCount);
  blasStatus = hipblasDgetriBatched(blasHandle, n, dAarray_const, lda, &P, dCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgetriBatched(cublasHandle_t handle, int n, const cuComplex* const A[], int lda, const int* P, cuComplex* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgetriBatched_v2(hipblasHandle_t handle, const int n, hipComplex* const A[], const int lda, int* ipiv, hipComplex* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgetriBatched_v2(blasHandle, n, complexAarray_const, lda, &P, complexCarray, ldc, &info, batchCount);
  blasStatus = hipblasCgetriBatched(blasHandle, n, complexAarray_const, lda, &P, complexCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgetriBatched(cublasHandle_t handle, int n, const cuDoubleComplex* const A[], int lda, const int* P, cuDoubleComplex* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgetriBatched_v2(hipblasHandle_t handle, const int n, hipDoubleComplex* const A[], const int lda, int* ipiv, hipDoubleComplex* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgetriBatched_v2(blasHandle, n, dcomplexAarray_const, lda, &P, dcomplexCarray, ldc, &info, batchCount);
  blasStatus = hipblasZgetriBatched(blasHandle, n, dcomplexAarray_const, lda, &P, dcomplexCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const float* const Aarray[], int lda, const int* devIpiv, float* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgetrsBatched(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, float* const A[], const int lda, const int* ipiv, float* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgetrsBatched(blasHandle, transa, n, nrhs, fAarray_const, lda, &P, fBarray, ldb, &info, batchCount);
  blasStatus = hipblasSgetrsBatched(blasHandle, transa, n, nrhs, fAarray_const, lda, &P, fBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const double* const Aarray[], int lda, const int* devIpiv, double* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgetrsBatched(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, double* const A[], const int lda, const int* ipiv, double* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgetrsBatched(blasHandle, transa, n, nrhs, dAarray_const, lda, &P, dBarray, ldb, &info, batchCount);
  blasStatus = hipblasDgetrsBatched(blasHandle, transa, n, nrhs, dAarray_const, lda, &P, dBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuComplex* const Aarray[], int lda, const int* devIpiv, cuComplex* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgetrsBatched_v2(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, hipComplex* const A[], const int lda, const int* ipiv, hipComplex* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgetrsBatched_v2(blasHandle, transa, n, nrhs, complexAarray_const, lda, &P, complexBarray, ldb, &info, batchCount);
  blasStatus = hipblasCgetrsBatched(blasHandle, transa, n, nrhs, complexAarray_const, lda, &P, complexBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuDoubleComplex* const Aarray[], int lda, const int* devIpiv, cuDoubleComplex* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgetrsBatched_v2(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, hipDoubleComplex* const A[], const int lda, const int* ipiv, hipDoubleComplex* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgetrsBatched_v2(blasHandle, transa, n, nrhs, dcomplexAarray_const, lda, &P, dcomplexBarray, ldb, &info, batchCount);
  blasStatus = hipblasZgetrsBatched(blasHandle, transa, n, nrhs, dcomplexAarray_const, lda, &P, dcomplexBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* const A[], int lda, float* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsmBatched(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const float* alpha, float* const AP[], int lda, float* BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasStrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);
  blasStatus = hipblasStrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* const A[], int lda, double* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsmBatched(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const double* alpha, double* const AP[], int lda, double* BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasDtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);
  blasStatus = hipblasDtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* const A[], int lda, cuComplex* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsmBatched_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA,hipblasDiagType_t diag, int m, int n, const hipComplex* alpha, const hipComplex* const AP[], int lda, hipComplex* const BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasCtrsmBatched_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);
  blasStatus = hipblasCtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* const A[], int lda, cuDoubleComplex* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsmBatched_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* const AP[], int lda, hipDoubleComplex* const BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasZtrsmBatched_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);
  blasStatus = hipblasZtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgeqrfBatched(cublasHandle_t handle, int m, int n, float* const Aarray[], int lda, float* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgeqrfBatched(hipblasHandle_t handle, const int m, const int n, float* const A[], const int lda, float* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgeqrfBatched(blasHandle, m, n, fAarray, lda, fTauarray, &info, batchCount);
  blasStatus = hipblasSgeqrfBatched(blasHandle, m, n, fAarray, lda, fTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgeqrfBatched(cublasHandle_t handle, int m, int n, double* const Aarray[], int lda, double* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgeqrfBatched(hipblasHandle_t handle, const int m, const int n, double* const A[], const int lda, double* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgeqrfBatched(blasHandle, m, n, dAarray, lda, dTauarray, &info, batchCount);
  blasStatus = hipblasDgeqrfBatched(blasHandle, m, n, dAarray, lda, dTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeqrfBatched(cublasHandle_t handle, int m, int n, cuComplex* const Aarray[], int lda, cuComplex* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgeqrfBatched_v2(hipblasHandle_t handle, const int m, const int n, hipComplex* const A[], const int lda, hipComplex* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgeqrfBatched_v2(blasHandle, m, n, complexAarray, lda, complexTauarray, &info, batchCount);
  blasStatus = hipblasCgeqrfBatched(blasHandle, m, n, complexAarray, lda, complexTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeqrfBatched(cublasHandle_t handle, int m, int n, cuDoubleComplex* const Aarray[], int lda, cuDoubleComplex* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgeqrfBatched_v2(hipblasHandle_t handle, const int m, const int n, hipDoubleComplex* const A[], const int lda, hipDoubleComplex* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgeqrfBatched_v2(blasHandle, m, n, dcomplexAarray, lda, dcomplexTauarray, &info, batchCount);
  blasStatus = hipblasZgeqrfBatched(blasHandle, m, n, dcomplexAarray, lda, dcomplexTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const float* A, int lda, const float* x, int incx, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSdgmm(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const float* AP, int lda, const float* x, int incx, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);
  blasStatus = hipblasSdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const double* A, int lda, const double* x, int incx, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDdgmm(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const double* AP, int lda, const double* x, int incx, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDdgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);
  blasStatus = hipblasDdgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdgmm_v2(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const hipComplex* AP, int lda, const hipComplex* x, int incx, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCdgmm_v2(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);
  blasStatus = hipblasCdgmm(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdgmm_v2(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZdgmm_v2(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);
  blasStatus = hipblasZdgmm(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);

  int deviceInfo = 0;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, float* const Aarray[], int lda, float* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgelsBatched(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, float* const A[], const int lda, float* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasSgelsBatched(blasHandle, blasOperation, m, n, nrhs, fAarray, lda, fCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasSgelsBatched(blasHandle, blasOperation, m, n, nrhs, fAarray, lda, fCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, double* const Aarray[], int lda, double* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgelsBatched(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, double* const A[], const int lda, double* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasDgelsBatched(blasHandle, blasOperation, m, n, nrhs, dAarray, lda, dCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasDgelsBatched(blasHandle, blasOperation, m, n, nrhs, dAarray, lda, dCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, cuComplex* const Aarray[], int lda, cuComplex* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgelsBatched_v2(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, hipComplex* const A[], const int lda, hipComplex* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasCgelsBatched_v2(blasHandle, blasOperation, m, n, nrhs, complexAarray, lda, complexCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasCgelsBatched(blasHandle, blasOperation, m, n, nrhs, complexAarray, lda, complexCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, cuDoubleComplex* const Aarray[], int lda, cuDoubleComplex* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgelsBatched_v2(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, hipDoubleComplex* const A[], const int lda, hipDoubleComplex* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasZgelsBatched_v2(blasHandle, blasOperation, m, n, nrhs, dcomplexAarray, lda, dcomplexCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasZgelsBatched(blasHandle, blasOperation, m, n, nrhs, dcomplexAarray, lda, dcomplexCarray, ldc, &info, &deviceInfo, batchCount);

  // NOTE: void CUBLASWINAPI cublasStrmm(char side, char uplo, char transa, char diag, int m, int n, float alpha, const float* A, int lda, float* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrmm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, float* C, int ldc);
  // CHECK: blasStatus = hipblasStrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);
  blasStatus = hipblasStrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);

  // NOTE: void CUBLASWINAPI cublasDtrmm(char side, char uplo, char transa, char diag, int m, int n, double alpha, const double* A, int lda, double* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrmm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, double* C, int ldc);
  // CHECK: blasStatus = hipblasDtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);

  // NOTE: void CUBLASWINAPI cublasCtrmm(char side, char uplo, char transa, char diag, int m, int n, cuComplex alpha, const cuComplex* A, int lda, cuComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrmm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipComplex* alpha, const hipComplex* A, int lda, const hipComplex* B, int ldb, hipComplex* C, int ldc);
  // CHECK: blasStatus = hipblasCtrmm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZtrmm(char side, char uplo, char transa, char diag, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrmm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* A, int lda, const hipDoubleComplex* B, int ldb, hipDoubleComplex* C, int ldc);
  // CHECK: blasStatus = hipblasZtrmm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);

  long long int strideA = 0;
  long long int strideB = 0;
  long long int strideC = 0;
  long long int stridex = 0;
  long long int stridey = 0;

#if CUDA_VERSION >= 7050
  // CHECK: __half* ha = 0;
  __half* ha = 0;
  // CHECK: __half* hA = 0;
  __half* hA = 0;
  // CHECK: __half* hb = 0;
  __half* hb = 0;
  // CHECK: __half* hB = 0;
  __half* hB = 0;
  // CHECK: __half* hc = 0;
  __half* hc = 0;
  // CHECK: __half* hC = 0;
  __half* hC = 0;

  // CHECK: __half** hAarray = 0;
  __half** hAarray = 0;
  // CHECK: const __half** const hAarray_const = const_cast<const __half**>(hAarray);
  const __half** const hAarray_const = const_cast<const __half**>(hAarray);
  // CHECK: __half** hBarray = 0;
  __half** hBarray = 0;
  // CHECK: const __half** const hBarray_const = const_cast<const __half**>(hBarray);
  const __half** const hBarray_const = const_cast<const __half**>(hBarray);
  // CHECK: __half** hCarray = 0;
  __half** hCarray = 0;
  // CHECK: const __half** const hCarray_const = const_cast<const __half**>(hCarray);
  const __half** const hCarray_const = const_cast<const __half**>(hCarray);
  // CHECK: __half** hxarray = 0;
  __half** hxarray = 0;
  // CHECK: const __half** const hxarray_const = const_cast<const __half**>(hxarray_const);
  const __half** const hxarray_const = const_cast<const __half**>(hxarray_const);
  // CHECK: __half** hyarray = 0;
  __half** hyarray = 0;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemm(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, const __half* B, int ldb, const __half* beta, __half* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasHgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasHalf* alpha, const hipblasHalf* AP, int lda, const hipblasHalf* BP, int ldb, const hipblasHalf* beta, hipblasHalf* CP, int ldc);
  // CHECK: blasStatus = hipblasHgemm(blasHandle, transa, transb, m, n, k, ha, hA, lda, hB, ldb, hb, hC, ldc);
  blasStatus = hipblasHgemm(blasHandle, transa, transb, m, n, k, ha, hA, lda, hB, ldb, hb, hC, ldc);
#endif

#if CUDA_VERSION >= 8000
  // CHECK: hipDataType DataType;
  // CHECK-NEXT: hipDataType DataType_t;
  // CHECK-NEXT: hipDataType blasDataType;
  // CHECK-NEXT: hipDataType R_16F = HIP_R_16F;
  // CHECK-NEXT: hipDataType C_16F = HIP_C_16F;
  // CHECK-NEXT: hipDataType R_32F = HIP_R_32F;
  // CHECK-NEXT: hipDataType C_32F = HIP_C_32F;
  // CHECK-NEXT: hipDataType R_64F = HIP_R_64F;
  // CHECK-NEXT: hipDataType C_64F = HIP_C_64F;
  // CHECK-NEXT: hipDataType R_8I = HIP_R_8I;
  // CHECK-NEXT: hipDataType C_8I = HIP_C_8I;
  // CHECK-NEXT: hipDataType R_8U = HIP_R_8U;
  // CHECK-NEXT: hipDataType C_8U = HIP_C_8U;
  // CHECK-NEXT: hipDataType R_32I = HIP_R_32I;
  // CHECK-NEXT: hipDataType C_32I = HIP_C_32I;
  // CHECK-NEXT: hipDataType R_32U = HIP_R_32U;
  // CHECK-NEXT: hipDataType C_32U = HIP_C_32U;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;

  // CHECK: hipDataType DataType_2, DataType_3;
  hipDataType DataType_2, DataType_3;

  // CHECK: hipblasGemmAlgo_t blasGemmAlgo;
  // CHECK-NEXT: hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasNrm2Ex(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, void* result, cudaDataType resultType, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasNrm2Ex_v2(hipblasHandle_t handle, int n, const void* x, hipDataType xType, int incx, void* result, hipDataType resultType, hipDataType executionType);
  // CHECK: blasStatus = hipblasNrm2Ex_v2(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);
  blasStatus = hipblasNrm2Ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, long long int strideA, const float* B, int ldb, long long int strideB, const float* beta, float* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const float* alpha, const float* AP, int lda, long long strideA, const float* BP, int ldb, long long strideB, const float* beta, float* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasSgemmStridedBatched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);
  blasStatus = hipblasSgemmStridedBatched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, long long int strideA, const double* B, int ldb, long long int strideB, const double* beta, double* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const double* alpha, const double* AP, int lda, long long strideA, const double* BP, int ldb, long long strideB, const double* beta, double* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasDgemmStridedBatched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);
  blasStatus = hipblasDgemmStridedBatched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, long long int strideA, const cuComplex* B, int ldb, long long int strideB, const cuComplex* beta, cuComplex* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemmStridedBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, long long strideA, const hipComplex* BP, int ldb, long long strideB, const hipComplex* beta, hipComplex* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasCgemmStridedBatched_v2(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);
  blasStatus = hipblasCgemmStridedBatched(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, long long int strideA, const cuDoubleComplex* B, int ldb, long long int strideB, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemmStridedBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, long long strideA, const hipDoubleComplex* BP, int ldb, long long strideB, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasZgemmStridedBatched_v2(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);
  blasStatus = hipblasZgemmStridedBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, long long int strideA, const __half* B, int ldb, long long int strideB, const __half* beta, __half* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasHgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasHalf* alpha, const hipblasHalf* AP, int lda, long long strideA, const hipblasHalf* BP, int ldb, long long strideB, const hipblasHalf* beta, hipblasHalf* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasHgemmStridedBatched(blasHandle, transa, transb, m, n, k, ha, hA, lda, strideA, hB, ldb, strideB, hb, hC, ldc, strideC, batchCount);
  blasStatus = hipblasHgemmStridedBatched(blasHandle, transa, transb, m, n, k, ha, hA, lda, strideA, hB, ldb, strideB, hb, hC, ldc, strideC, batchCount);

  void* aptr = nullptr;
  void* Aptr = nullptr;
  void* bptr = nullptr;
  void* Bptr = nullptr;
  void* cptr = nullptr;
  void* Cptr = nullptr;
  void* xptr = nullptr;
  void* yptr = nullptr;
  void* sptr = nullptr;

  // CHECK: hipDataType Atype;
  // CHECK-NEXT: hipDataType Btype;
  // CHECK-NEXT: hipDataType Ctype;
  // CHECK-NEXT: hipDataType Xtype;
  // CHECK-NEXT: hipDataType Ytype;
  // CHECK-NEXT: hipDataType CStype;
  // CHECK-NEXT: hipDataType Executiontype;
  hipDataType Atype;
  hipDataType Btype;
  hipDataType Ctype;
  hipDataType Xtype;
  hipDataType Ytype;
  hipDataType CStype;
  hipDataType Executiontype;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScalEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, void* x, cudaDataType xType, int incx, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScalEx_v2(hipblasHandle_t handle, int n, const void* alpha, hipDataType alphaType, void* x, hipDataType xType, int incx, hipDataType executionType);
  // CHECK: blasStatus = hipblasScalEx_v2(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);
  blasStatus = hipblasScalEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasAxpyEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, const void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, cudaDataType executiontype);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasAxpyEx_v2(hipblasHandle_t handle, int n, const void* alpha, hipDataType alphaType, const void* x, hipDataType xType, int incx, void* y, hipDataType yType, int incy, hipDataType executionType);
  // CHECK: blasStatus = hipblasAxpyEx_v2(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);
  blasStatus = hipblasAxpyEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDotEx_v2(hipblasHandle_t handle, int n, const void* x, hipDataType xType, int incx, const void* y, hipDataType yType, int incy, void* result, hipDataType resultType, hipDataType executionType);
  // CHECK: blasStatus = hipblasDotEx_v2(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotcEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDotcEx_v2(hipblasHandle_t handle, int n, const void* x, hipDataType xType, int incx, const void* y, hipDataType yType, int incy, void* result, hipDataType resultType, hipDataType executionType);
  // CHECK: blasStatus = hipblasDotcEx_v2(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotcEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
#endif

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 11000
  // CHECK: hipDataType computeType;
  cudaDataType computeType;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, const void* B, cudaDataType Btype, int ldb, const void* beta, void* C, cudaDataType Ctype, int ldc, cudaDataType computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipDataType aType, int lda, const void* B, hipDataType bType, int ldb, const void* beta, void* C, hipDataType cType, int ldc, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmEx_v2(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
  blasStatus = cublasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* const Aarray[], int lda, const __half* const Barray[], int ldb, const __half* beta, __half* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasHgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasHalf* alpha, const hipblasHalf* const AP[], int lda, const hipblasHalf* const BP[], int ldb, const hipblasHalf* beta, hipblasHalf* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasHgemmBatched(blasHandle, transa, transb, m, n, k, ha, hAarray_const, lda, hBarray_const, ldb, hb, hCarray, ldc, batchCount);
  blasStatus = hipblasHgemmBatched(blasHandle, transa, transb, m, n, k, ha, hAarray_const, lda, hBarray_const, ldb, hb, hCarray, ldc, batchCount);

  // CHECK: hipblasMath_t blasMath;
  // CHECK-NEXT: hipblasMath_t BLAS_DEFAULT_MATH = HIPBLAS_DEFAULT_MATH;
  // CHECK-NEXT: hipblasMath_t BLAS_TENSOR_OP_MATH = HIPBLAS_TENSOR_OP_MATH;
  hipblasMath_t blasMath;
  hipblasMath_t BLAS_DEFAULT_MATH = HIPBLAS_DEFAULT_MATH;
  hipblasMath_t BLAS_TENSOR_OP_MATH = HIPBLAS_TENSOR_OP_MATH;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetMathMode(cublasHandle_t handle, cublasMath_t* mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetMathMode(hipblasHandle_t handle, hipblasMath_t* mode);
  // CHECK: blasStatus = hipblasGetMathMode(blasHandle, &blasMath);
  blasStatus = hipblasGetMathMode(blasHandle, &blasMath);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetMathMode(cublasHandle_t handle, cublasMath_t mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetMathMode(hipblasHandle_t handle, hipblasMath_t mode);
  // CHECK: blasStatus = hipblasSetMathMode(blasHandle, blasMath);
  blasStatus = hipblasSetMathMode(blasHandle, blasMath);
#endif

#if CUDA_VERSION >= 9010 && CUDA_VERSION < 11000
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* const Aarray[], cudaDataType Atype, int lda, const void* const Barray[], cudaDataType Btype, int ldb, const void* beta, void* const Carray[], cudaDataType Ctype, int ldc, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmBatchedEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A[], hipDataType aType, int lda, const void* B[], hipDataType bType, int ldb, const void* beta, void* C[], hipDataType cType, int ldc, int batchCount, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmBatchedEx_v2(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmStridedBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, long long int strideA, const void* B, cudaDataType Btype, int ldb, long long int strideB, const void* beta, void* C, cudaDataType Ctype, int ldc, long long int strideC, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmStridedBatchedEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipDataType aType, int lda, hipblasStride strideA, const void* B, hipDataType bType, int ldb, hipblasStride strideB, const void* beta, void* C, hipDataType cType, int ldc, hipblasStride strideC, int batchCount, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmStridedBatchedEx_v2(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasRotEx(cublasHandle_t handle, int n, void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, const void* c, const void* s, cudaDataType csType, cudaDataType executiontype);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasRotEx_v2(hipblasHandle_t handle, int n, void* x, hipDataType xType, int incx, void* y, hipDataType yType, int incy, const void* c, const void* s, hipDataType csType, hipDataType executionType);
  // CHECK: blasStatus = hipblasRotEx_v2(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
  blasStatus = hipblasRotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipDataType R_16BF = HIP_R_16BF;
  // CHECK-NEXT: hipDataType C_16BF = HIP_C_16BF;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;

  // CHECK: hipblasMath_t BLAS_PEDANTIC_MATH = HIPBLAS_PEDANTIC_MATH;
  // CHECK-NEXT: hipblasMath_t BLAS_TF32_TENSOR_OP_MATH = HIPBLAS_TF32_TENSOR_OP_MATH;
  // CHECK-NEXT: hipblasMath_t BLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION = HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION;
  hipblasMath_t BLAS_PEDANTIC_MATH = HIPBLAS_PEDANTIC_MATH;
  hipblasMath_t BLAS_TF32_TENSOR_OP_MATH = HIPBLAS_TF32_TENSOR_OP_MATH;
  hipblasMath_t BLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION = HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION;

  // CHECK: hip_bfloat16** bf16Aarray = 0;
  __hip_bfloat16** bf16Aarray = 0;
  // CHECK: const hip_bfloat16** const bf16Aarray_const = const_cast<const hip_bfloat16**>(bf16Aarray);
  const __hip_bfloat16** const bf16Aarray_const = const_cast<const __hip_bfloat16**>(bf16Aarray);
  // CHECK: hip_bfloat16** bf16xarray = 0;
  __hip_bfloat16** bf16xarray = 0;
  // CHECK: const hip_bfloat16** const bf16xarray_const = const_cast<const hip_bfloat16**>(bf16xarray_const);
  const __hip_bfloat16** const bf16xarray_const = const_cast<const __hip_bfloat16**>(bf16xarray_const);
  // CHECK: hip_bfloat16** bf16yarray = 0;
  __hip_bfloat16** bf16yarray = 0;

  // CHECK: hipblasComputeType_t blasComputeType;
  hipblasComputeType_t blasComputeType;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, const void* B, cudaDataType Btype, int ldb, const void* beta, void* C, cudaDataType Ctype, int ldc, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipDataType aType, int lda, const void* B, hipDataType bType, int ldb, const void* beta, void* C, hipDataType cType, int ldc, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmEx_v2(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, blasComputeType, blasGemmAlgo);
  blasStatus = hipblasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, blasComputeType, blasGemmAlgo);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* const Aarray[], cudaDataType Atype, int lda, const void* const Barray[], cudaDataType Btype, int ldb, const void* beta, void* const Carray[], cudaDataType Ctype, int ldc, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmBatchedEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A[], hipDataType aType, int lda, const void* B[], hipDataType bType, int ldb, const void* beta, void* C[], hipDataType cType, int ldc, int batchCount, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmBatchedEx_v2(blasHandle, transa, transb, m, n, k, aptr, voidAarray, Atype, lda, voidBarray, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, blasComputeType, blasGemmAlgo);
  blasStatus = hipblasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray, Atype, lda, voidBarray, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, blasComputeType, blasGemmAlgo);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmStridedBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, long long int strideA, const void* B, cudaDataType Btype, int ldb, long long int strideB, const void* beta, void* C, cudaDataType Ctype, int ldc, long long int strideC, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmStridedBatchedEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipDataType aType, int lda, hipblasStride strideA, const void* B, hipDataType bType, int ldb, hipblasStride strideB, const void* beta, void* C, hipDataType cType, int ldc, hipblasStride strideC, int batchCount, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmStridedBatchedEx_v2(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, blasComputeType, blasGemmAlgo);
  blasStatus = hipblasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, blasComputeType, blasGemmAlgo);
#endif

#if CUDA_VERSION > 11060 && CUBLAS_VERSION >= 110902 // CUDA 11.6.2
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemvBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuComplex* alpha, const cuComplex* const Aarray[], int lda, const cuComplex* const xarray[], int incx, const cuComplex* beta, cuComplex* const yarray[], int incy, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemvBatched_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipComplex* alpha, const hipComplex* const AP[], int lda, const hipComplex* const x[], int incx, const hipComplex* beta, hipComplex* const y[], int incy, int batchCount);
  // CHECK: blasStatus = hipblasCgemvBatched_v2(blasHandle, blasOperation, m, n, &complexa, complexAarray_const, lda, complexXarray_const, incx, &complexb, complexYarray, incy, batchCount);
  blasStatus = hipblasCgemvBatched(blasHandle, blasOperation, m, n, &complexa, complexAarray_const, lda, complexXarray_const, incx, &complexb, complexYarray, incy, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemvBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* const Aarray[], int lda, const cuDoubleComplex* const xarray[], int incx, const cuDoubleComplex* beta, cuDoubleComplex* const yarray[], int incy, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemvBatched_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* const A[], int lda, const hipDoubleComplex* const x[], int incx, const hipDoubleComplex* beta, hipDoubleComplex* const y[], int incy, int batchCount);
  // CHECK: blasStatus = hipblasZgemvBatched_v2(blasHandle, blasOperation, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexXarray_const, incx, &dcomplexb, dcomplexYarray, incy, batchCount);
  blasStatus = hipblasZgemvBatched(blasHandle, blasOperation, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexXarray_const, incx, &dcomplexb, dcomplexYarray, incy, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemvStridedBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, long long int strideA, const cuComplex* x, int incx, long long int stridex, const cuComplex* beta, cuComplex* y, int incy, long long int stridey, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemvStridedBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, int m, int n, const hipComplex* alpha, const hipComplex* AP, int lda, hipblasStride strideA, const hipComplex* x, int incx, hipblasStride stridex, const hipComplex* beta, hipComplex* y, int incy, hipblasStride stridey, int batchCount);
  // CHECK: blasStatus = hipblasCgemvStridedBatched_v2(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, strideA, &complexx, incx, stridex, &complexb, &complexy, incy, stridey, batchCount);
  blasStatus = hipblasCgemvStridedBatched(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, strideA, &complexx, incx, stridex, &complexb, &complexy, incy, stridey, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemvStridedBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, long long int strideA, const cuDoubleComplex* x, int incx, long long int stridex, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy, long long int stridey, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemvStridedBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, hipblasStride strideA, const hipDoubleComplex* x, int incx, hipblasStride stridex, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy, hipblasStride stridey, int batchCount);
  // CHECK: blasStatus = hipblasZgemvStridedBatched_v2(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexx, incx, stridex, &dcomplexb, &dcomplexy, incy, stridey, batchCount);
  blasStatus = hipblasZgemvStridedBatched(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexx, incx, stridex, &dcomplexb, &dcomplexy, incy, stridey, batchCount);
#endif

  return 0;
}
